/*
Compile:
    
    make cuda

Usage (with job.sh):

    make sub
    
3D viewer:

    make physicsViewer
    ./physicsViewer output_file
    
Jan Mas Rovira
Andrés Mingorance López
Albert Puente Encinas
*/

#include <stdio.h>  // e.g. printf
#include <stdlib.h> // e.g. malloc, RAND_MAX, exit
#include <math.h>   // e.g. sin, abs
#include <sys/time.h>
#include <hip/hip_runtime.h>

// Algorithm parameters
#define N 1024*8 // Si incrementa FALLA (+110 MB)!
#define ITERATIONS 2048
#define G 9.81
#define BOUNCE_DECAY 0.5
#define GLOBAL_DECAY 0.004
#define POINT_RADIUS 0.3
#define TIME_SPEED 0.0075
#define MAX_TRIES 1e4
#define SEED 27

#define DUMP_RATIO 2

// CUDA Variables
#define nThreads 256
#define nBlocks N/nThreads

// c++ style
#define bool int
#define true 1
#define false 0

#define WALLS true
#define DUMP true

// Timers
unsigned long long initialGenTime;
unsigned long long interactionsTime;
unsigned long long worldInteractionsTime;
unsigned long long gravityTime;
unsigned long long advanceTime;
unsigned long long frameTime;
unsigned long long totalTime;

inline void tic(unsigned long long* time) {
    struct timeval t;
    gettimeofday(&t, NULL);
    *time = t.tv_sec*1000000 + t.tv_usec - *time;
    
}
#define toc tic


typedef struct {
    float x, y, z;
} Vector;

typedef struct {
    float x, y, z;      // Position
    Vector velocity;    // Velocity
} Point;

typedef struct {
    Point points[N];
} PointSet;

void checkCudaError(char msg[]) {
    hipError_t error;
    error = hipGetLastError();
    if (error) {
        printf("Error: %s: %s\n", msg, hipGetErrorString(error));
        exit(1);
    }
}

inline float dist(Point* a, Point* b) {
    return sqrt(pow(a->x - b->x, 2)+pow(a->y - b->y, 2)+pow(a->z - b->z, 2));
}

__device__ inline float gpu_dist(Point* a, Point* b) {
    return sqrt(pow(a->x - b->x, 2)+pow(a->y - b->y, 2)+pow(a->z - b->z, 2));
}

__device__ inline float distNext(Point* a, Point* b) {
    return sqrt( pow(a->x + a->velocity.x*TIME_SPEED - (b->x + b->velocity.x*TIME_SPEED), 2)+
                 pow(a->y + a->velocity.y*TIME_SPEED - (b->y + b->velocity.y*TIME_SPEED), 2)+
                 pow(a->z + a->velocity.z*TIME_SPEED - (b->z + b->velocity.z*TIME_SPEED), 2));
}

bool collides(Point* p, PointSet* PS, int from, int to) {
    for (int i = from; i < to; ++i) {
        if (dist(p, &PS->points[i]) < POINT_RADIUS*2) {
            return true;
        }
    }
    return false;
}

__device__ Vector diffVector(Point* a, Point* b) {
    Vector v;
    v.x = a->x - b->x;
    v.y = a->y - b->y;
    v.z = a->z - b->z;
    return v;
}

__device__ inline float dotProduct(Vector a, Vector b) {
    return a.x*b.x + a.y*b.y + a.z*b.z;
}


__global__ void kernel_interaction(PointSet* P, PointSet* Q) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;    
    
    int j = 1 + floor(-0.5 + sqrt(0.25 + 2 * index));
    int triangularNumber = j * (j - 1) / 2;
    int i = index - triangularNumber;
    
    Point* a = &P->points[i];
    Point* b = &P->points[j];

    float distance = gpu_dist(a, b);
    
    if (distance > 2*POINT_RADIUS + 0.05) return;
    
    if (distance == 0) return; // AVOID NAN, PROVISIONAL
    
    if (distance < distNext(a, b)) return;
    
    Point* aq = &Q->points[i];
    Point* bq = &Q->points[j];
    
    // Get the components of the velocity vectors which are parallel to the collision.
    // The perpendicular component remains the same for both fish
    Vector collision = diffVector(a, b);
    
    //
    //distance = 2*POINT_RADIUS;
    collision.x /= distance;
    collision.y /= distance;
    collision.z /= distance;
    
    float aci = dotProduct(collision, a->velocity); 
    float bci = dotProduct(collision, b->velocity); 

    // Replace the collision velocity components with the new ones

    atomicAdd(&aq->velocity.x, (bci - aci) * collision.x);
    atomicAdd(&aq->velocity.y, (bci - aci) * collision.y);
    atomicAdd(&aq->velocity.z, (bci - aci) * collision.z);
        
    atomicAdd(&bq->velocity.x, (aci - bci) * collision.x);
    atomicAdd(&bq->velocity.y, (aci - bci) * collision.y);
    atomicAdd(&bq->velocity.z, (aci - bci) * collision.z);
}

void computeInteraction(PointSet* gpu_P, PointSet* gpu_Q) {   
    /*
     * (n*(n-1))/2 elements
     * because we are emulating the following loop
     * 
     * for (int i = 0; i < N; ++i)
     *     for (int j = i + 1; j < N; ++j)
     *         kernel_interaction(i, j);
     */
    // nThreads is 1024
    
    int nElem = (N*(N-1))/2;
    int nElemBlocks = nElem/nThreads;
    kernel_interaction<<<nElemBlocks, nThreads>>>(gpu_P, gpu_Q);
    checkCudaError((char *) "kernel call in interaction");    
    hipDeviceSynchronize();
}

__global__ void kernel_gravity(PointSet* P) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    P->points[id].velocity.y -= G*TIME_SPEED;    
}

void applyGravity(PointSet* gpu_P) {
    kernel_gravity<<<nBlocks, nThreads>>>(gpu_P);
    checkCudaError((char *) "kernel call in applyGravity");    
    hipDeviceSynchronize();
}

__global__ void kernel_advance(PointSet* P, PointSet* Q) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    Point* p = &P->points[id];
    p->x += p->velocity.x*TIME_SPEED;
    p->y += p->velocity.y*TIME_SPEED;
    p->z += p->velocity.z*TIME_SPEED;
    p->velocity.x *= (1-GLOBAL_DECAY);
    p->velocity.y *= (1-GLOBAL_DECAY);
    p->velocity.z *= (1-GLOBAL_DECAY);
    
    Q->points[id] = *p;
}

void advanceAndCopy(PointSet* gpu_P, PointSet* gpu_Q) {
    kernel_advance<<<nBlocks, nThreads>>>(gpu_P, gpu_Q);
    checkCudaError((char *) "kernel call in advance");    
    hipDeviceSynchronize();
}

__device__ inline void ifelse(bool condition, float* dest, float a, float b) {
    *dest = condition*a + !condition*b;    
}

__global__ void kernel_world(PointSet* P) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    Point* p = &P->points[id];
    if (p->y < POINT_RADIUS) {
        p->y = POINT_RADIUS;
        p->velocity.y = abs(p->velocity.y) * (1.0 - BOUNCE_DECAY);
    }
    
    if (WALLS) { // 4 walls x = -10, 10 and z = -10, 10
        if (p->x < -10.0 + POINT_RADIUS) {
            p->x = -10 + POINT_RADIUS;
            p->velocity.x = abs(p->velocity.x) * (1.0 - BOUNCE_DECAY);
        }
        else if (p->x > 10.0 - POINT_RADIUS) {
            p->x = 10 - POINT_RADIUS;
            p->velocity.x = -abs(p->velocity.x) * (1.0 - BOUNCE_DECAY);
        }            
        
        if (p->z < -10.0 + POINT_RADIUS) {
            p->z = -10 + POINT_RADIUS;
            p->velocity.z = abs(p->velocity.z) * (1.0 - BOUNCE_DECAY);
        }            
        else if (p->z > 10.0 - POINT_RADIUS) {
            p->z = 10 - POINT_RADIUS;
            p->velocity.z = -abs(p->velocity.z) * (1.0 - BOUNCE_DECAY);
        }
        
    }
}

void computeInteractionWorld(PointSet* gpu_P) {
    kernel_world<<<nBlocks, nThreads>>>(gpu_P);
    checkCudaError((char *) "kernel call in interactionWorld");    
    hipDeviceSynchronize();
}


void computePhysics(PointSet* gpu_P, PointSet* gpu_Q) {
    tic(&gravityTime);
    applyGravity(gpu_Q); 
    toc(&gravityTime);
    
    tic(&worldInteractionsTime);
    computeInteractionWorld(gpu_Q);
    tic(&worldInteractionsTime);
    
    tic(&interactionsTime);
    computeInteraction(gpu_P, gpu_Q);
    tic(&interactionsTime);
    
    tic(&advanceTime);
    advanceAndCopy(gpu_Q, gpu_P);
    toc(&advanceTime);
}

void generateInitialConfiguration(PointSet* gpu_P, PointSet* gpu_Q) {
    tic(&initialGenTime);
    PointSet* P = (PointSet*) malloc(sizeof(PointSet));
    
    for (int i = 0; i < N; ++i) {
        
        Point* p = &P->points[i]; 

        p->x = 12.0*(float)rand()/(float)(RAND_MAX) - 6.0;
        p->y = 400.0*(float)rand()/(float)(RAND_MAX) + 1.0;
        p->z = 12.0*(float)rand()/(float)(RAND_MAX) - 6.0;       
        
        p->velocity.x = 0.0;
        p->velocity.y = -3.5;
        p->velocity.z = 0.0;
                
        int tests = 0;
        while (tests < MAX_TRIES && collides(p, P, 0, i)) {

            p->x = 12.0*(float)rand()/(float)(RAND_MAX) - 6.0;
            p->y = 400.0*(float)rand()/(float)(RAND_MAX) + 1.0;
            p->z = 12.0*(float)rand()/(float)(RAND_MAX) - 6.0;       
            ++tests;
        }
        if (tests == MAX_TRIES) {
            printf("Error during the generation of the initial conf.\n");
            exit(1);
        }
    }
    
    hipMemcpy(gpu_P, P, sizeof(PointSet), hipMemcpyHostToDevice);
    checkCudaError((char *) "host -> gpu_P");
    
    hipMemcpy(gpu_Q, P, sizeof(PointSet), hipMemcpyHostToDevice);
    checkCudaError((char *) "host -> gpu_Q");
    
    hipDeviceSynchronize();
    
    toc(&initialGenTime);
}

void DUMPInitialParams() {
    printf("%i %i %f\n", N, ITERATIONS/DUMP_RATIO, POINT_RADIUS);
}

__global__ void kernel_print(PointSet* P) {
     int id = blockIdx.x * blockDim.x + threadIdx.x;
     printf("%f %f %f\n", P->points[id].x, P->points[id].y, P->points[id].z);     
}

void dump(PointSet* gpu_P) {
    
    kernel_print<<<nBlocks, nThreads>>>(gpu_P);
    checkCudaError((char *) "kernel call in interaction"); 
    
    hipDeviceSynchronize();

}

void initTimes() {
    initialGenTime = 0;
    interactionsTime = 0;
    worldInteractionsTime = 0;
    gravityTime = 0;
    advanceTime = 0;
    frameTime = 0;
    totalTime = 0;
}

void printTimes() {
    printf("CUDA physics algorithm has finished:\n");
    printf("    Init gen:     %f s.\n", (double)initialGenTime/1000000);
    printf("    Interactions: %f s.\n", (double)interactionsTime/1000000);
    printf("    World int.:   %f s.\n", (double)worldInteractionsTime/1000000);
    printf("    Gravity:      %f s.\n", (double)gravityTime/1000000);
    printf("    Advance:      %f s.\n", (double)advanceTime/1000000);
    printf("    Total time:   %f s.\n", (double)totalTime/1000000);
}

void cudaPhysics() {
    
    DUMPInitialParams();  
    
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 100*1024*1024); // 100 FIFO Limit MB
    
    PointSet* gpu_P;
    PointSet* gpu_Q;
    hipMalloc((void **) &gpu_P, sizeof(PointSet));
    checkCudaError((char *) "hipMalloc of P");
    hipMalloc((void **) &gpu_Q, sizeof(PointSet));
    checkCudaError((char *) "hipMalloc of Q");
    
    tic(&totalTime);
    srand(SEED);
    generateInitialConfiguration(gpu_P, gpu_Q); // *CPU_P = *gpu_P = *gpu_Q
    
    
    for (int i = 0; i < ITERATIONS; ++i) {
        tic(&frameTime);
        computePhysics(gpu_P, gpu_Q);      
        if (DUMP) {
            if (i%DUMP_RATIO == 0) dump(gpu_P);
        }
        else printf("IT %i\n", i);
        
        toc(&frameTime);    
    }
    
    toc(&totalTime);
    if (!DUMP) printTimes();   
}

int main(int argc, char** argv) {

    cudaPhysics();
    return 0;
}

